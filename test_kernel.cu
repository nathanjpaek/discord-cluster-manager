#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void saxpy_kernel(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

void saxpy(int n, float a, float *x, float *y) {
    float *d_x, *d_y;
    
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    
    saxpy_kernel<<<numBlocks, blockSize>>>(n, a, d_x, d_y);
    
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_x);
    hipFree(d_y);
}

int main(void) {
    int N = 1 << 20; // 1M elements
    float a = 2.0f;
    
    float *x = (float*)malloc(N * sizeof(float));
    float *y = (float*)malloc(N * sizeof(float));
    
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    
    saxpy(N, a, x, y);
    
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 4.0f));
    }
    printf("Max error: %f\n", maxError);
    
    free(x);
    free(y);
    
    return 0;
}
